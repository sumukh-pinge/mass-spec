#include "hip/hip_runtime.h"
#include "../include/kernels.cuh"
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

struct square { __host__ __device__ float operator()(float x) { return x * x; } };

__global__ void normMatRow(float* result, float* inputMat, int setNum, int colNum) {
    for (int rowNum = blockIdx.x * blockDim.x + threadIdx.x; 
        rowNum < setNum; 
        rowNum += blockDim.x * gridDim.x)
    {
        // result[rowNum] = normf(colNum, inputMat + rowNum * colNum);
        result[rowNum] = sqrt(thrust::transform_reduce(thrust::device, inputMat + rowNum * colNum, inputMat + (rowNum + 1) * colNum, square(), 0.0f, thrust::plus<float>()));
    }
}

__global__ void cosineSimilarityVec(float* result, float* norm_1, int colNum, float* norm_2_const, int dataidx) {
    float norm_2 = norm_2_const[dataidx];
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < colNum)
        result[idx] = result[idx] / (norm_1[idx] * norm_2);
}

__global__
void guessVecGenCompareCosine(int* y_pred, float* weights_norm, float* data_norm,
    const float * guess_table, const int setNum, const int n_class) {

    int rowNum = threadIdx.x + blockDim.x * blockIdx.x;
    float norm_2 = data_norm[rowNum];

    if (rowNum < setNum) {
        float max_value = guess_table[n_class * rowNum + 0] / (weights_norm[0] * norm_2);
        // float max_value = guess_table[n_class * rowNum + 0]; 
        int max_idx = 0;
        for (int j = 1; j < n_class; j++){
            float val_to_compare = guess_table[n_class * rowNum + j] / (weights_norm[j] * norm_2);
            if (max_value < val_to_compare) {
                max_value = val_to_compare;
                max_idx = j;
            }
        }
        y_pred[rowNum] = max_idx;
    }
}

__global__
void guessVecGenCompareDot(int* y_pred,
    const float * guess_table, const int setNum, const int n_class) {

    int rowNum = threadIdx.x + blockDim.x * blockIdx.x;
    if (rowNum < setNum) {
        float max_value = guess_table[n_class * rowNum + 0];
        // float max_value = guess_table[n_class * rowNum + 0];
        int max_idx = 0;
        for (int j = 1; j < n_class; j++){
            float val_to_compare = guess_table[n_class * rowNum + j];
            if (max_value < val_to_compare) {
                max_value = val_to_compare;
                max_idx = j;
            }
        }
        y_pred[rowNum] = max_idx;
    }
}

